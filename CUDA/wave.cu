
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void printfinal(void);

int nsteps, tpoints;
float  values[MAXPOINTS+2];

void check_param(void) {
	char tchar[20];
	while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
		printf("Enter number of points along vibrating string [%d-%d]: "
				,MINPOINTS, MAXPOINTS);
		scanf("%s", tchar);
		tpoints = atoi(tchar);
		if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
			printf("Invalid. Please enter value between %d and %d\n", 
					MINPOINTS, MAXPOINTS);
	}
	while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
		printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
		scanf("%s", tchar);
		nsteps = atoi(tchar);
		if ((nsteps < 1) || (nsteps > MAXSTEPS))
			printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
	}
	printf("Using points = %d, steps = %d\n", tpoints, nsteps);
}

__global__ void wave(float* oldval_d, float* values_d, float* newval_d, int nsteps, int tpoints) {
	int idx = threadIdx.x;
	float x, fac = 2.0 * PI, k = idx, tmp = tpoints - 1;
	x = k / tmp;
	values_d[idx] = sin(fac * x);
	float dtime = 0.3, c = 1.0, dx = 1.0;
	float tau = c * dtime / dx;
	float sqtau = tau * tau;
	oldval_d[idx] = values_d[idx];
	for (int i = 0; i < nsteps; i++) {
		newval_d[idx] = (2.0 * values_d[idx]) - oldval_d[idx] + (sqtau * (-2.0) * values_d[idx]);
		oldval_d[idx] = values_d[idx];
		values_d[idx] = newval_d[idx];
	}
}

void printfinal() {
	for (int i = 0; i < tpoints; i++) {
		printf("%6.4f ", values[i]);
		if (i % 10 == 9)
			printf("\n");
	}
}

int main(int argc, char *argv[]) {
	float *oldval_d, *values_d, *newval_d;
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();
	printf("Initializing points on the line...\n");
	printf("Updating all points for all time steps...\n");
	hipMalloc((void**)&oldval_d, sizeof(float) * tpoints);
	hipMalloc((void**)&values_d, sizeof(float) * tpoints);
	hipMalloc((void**)&newval_d, sizeof(float) * tpoints);
	wave<<<1, tpoints>>>(oldval_d, values_d, newval_d, nsteps, tpoints);
	hipMemcpy(values, values_d, sizeof(float) * tpoints, hipMemcpyDeviceToHost);
	hipFree(oldval_d);
	hipFree(values_d);
	hipFree(newval_d);
	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
	return 0;
}
